#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define CUDA_CHECK(X)                                                     \
    {                                                                     \
        hipError_t _m_cudaStat = X;                                      \
        if (hipSuccess != _m_cudaStat) {                                 \
            fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
                    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#ifndef SIZE
#define SIZE 8192 * 12 * 12
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 256
#endif

#ifndef MAX_DIGIT
#define MAX_DIGIT 9999
#endif

#ifndef GIPS
#define GIPS 0
#endif

#ifndef TEST
#define TEST 0
#endif

#define GRIDSIZE ((SIZE - 1) / BLOCKSIZE + 1)
#define RADIX 10
#define MAXSM 12
#define FILE_TO_OPEN "Streams_Texture_measure.csv"

texture<int, 1> texture_semiSortArray;
/**
 * This device function is used to retrive values from the array
 * saved in the texture memory
 * */
__device__ float fetch_radixArrayElement(int value) {
    return tex1Dfetch(texture_semiSortArray, value);
}
/**
 *  This kernel will be launched on GRIDSIZE * BLOCKSIZE threads, in order to copy the values of the semiSortArray in inArray
 *
 * **/
__global__ void copyKernel(int *inArray, int offsette, int arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < arrayLength) {
        inArray[index] = fetch_radixArrayElement(index + offsette);
    }
}
/**
 * This kernel will be launched on GRIDSIZE * BLOCKSIZE threads, so that each thread will calculate its local maximum and minimum value
 *
 * */
__global__ void reduceMaxMin(int *g_idata, int *g_maxdata, int *g_mindata) {
    __shared__ int smaxdata[(SIZE / GRIDSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(SIZE / GRIDSIZE)];  // each thread loads one element from global to shared mem unsigned
    int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    smaxdata[tid] = g_idata[i];
    smindata[tid] = g_idata[i];
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
    }

    // write result for this block to global mem
    if (tid == 0) {
        g_maxdata[blockIdx.x] = smaxdata[0];
        g_mindata[blockIdx.x] = smindata[0];
    }
}
/**
 * This kernel will be launched on 1 * BLOCKSIZE threads, so that a global
 * maximum and minimum value shared between all blocks will be calcualted
 * */
__global__ void reduceMaxMin_Service(int *g_maxdata, int *g_mindata, int *max, int *min) {
    __shared__ int smaxdata[(BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(BLOCKSIZE)];
    int tid = threadIdx.x;
    smaxdata[tid] = g_maxdata[tid];
    smindata[tid] = g_mindata[tid];
    for (unsigned int s = 1; s < GRIDSIZE / BLOCKSIZE; s++) {
        int index = BLOCKSIZE * s + tid;
        if (smaxdata[tid] < g_maxdata[index])
            smaxdata[tid] = g_maxdata[index];
        if (smindata[tid] > g_mindata[index])
            smindata[tid] = g_mindata[index];
    }
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
    }
    // write result for this block to global mem
    if (tid == 0) {
        *max = smaxdata[0];
        *min = smindata[0];
    }
}
/**
 * This kernel will be launched on on numbers of threads that depends on the length of the array
 * */
__global__ void resetBucket(int *bucket) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    bucket[index] = 0;
}
/**
 * This kernel will be launched on GRIDSIZE * BLOCKSIZE threads, so that each thread for
 * the specific significant digit of the assigned value will increase the frequencies of the digit
 * */
__global__ void histogramKernel(int *inArray, int *outArray, int *radixArray, int arrayLength, int significantDigit, int minElement) {
    __shared__ int inArrayShared[BLOCKSIZE];
    __shared__ int outArrayShared[RADIX];
    __shared__ int radixArrayShared[BLOCKSIZE];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread = threadIdx.x;
    int blockIndex = blockIdx.x * RADIX;
    if (index > arrayLength) {
        return;
    }
    int radix;
    int arrayElement;
    int i;

    if (thread < RADIX) {
        outArrayShared[thread] = 0;
    }
    __syncthreads();
    if (index < arrayLength) {
        inArrayShared[thread] = inArray[index];

        arrayElement = inArrayShared[thread] - minElement;
        radix = ((arrayElement / significantDigit) % 10);
        radixArrayShared[thread] = radix;
        atomicAdd(&outArrayShared[radix], 1);

        radixArray[index] = radixArrayShared[thread];
    }
    __syncthreads();

    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[blockIndex + i] += outArrayShared[i];
        }
    }
}
/**
 * This kernel will be launched on 1 * RADIX threads, so that the array containing the frequencies
 * for each block is addictioned to that of the other blocks.
 * Then the value in each position of the resulting array is addictioned with the value in the previus one position.
 * */
__global__ void combineBucket(int *blockBucketArray, int *bucketArray, int block_size) {
    __shared__ int bucketArrayShared[RADIX];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index] = 0;

    for (i = index; i < RADIX * block_size; i = i + RADIX) {
        atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 1; i < RADIX; i++) {
            bucketArrayShared[i] += bucketArrayShared[i - 1];
        }
    }
    atomicAdd(&bucketArray[index], bucketArrayShared[index]);
}
/**
 * This kernel will be launched on GRIDSIZE * BLOCKSIZE, so that a sorting
 * for the specific significantDigit between the numbers is made
 * */
__global__ void semiSortKernel(int *inArray, int *outArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;
    if (index < arrayLength) {
        arrayElement = inArray[index];
        arrayIndex = indexArray[index];
        outArray[arrayIndex] = arrayElement;
    }
}
/**
 * Print all the array
 * */
void printArray(int *array, int size) {
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}

/**
 * This functions makes the csv file with all the information needed
 * */
void make_csv(float time, float N) {
    FILE *fp;
    if (access(FILE_TO_OPEN, F_OK) == 0) {
        fp = fopen(FILE_TO_OPEN, "a");

    } else {
        fp = fopen(FILE_TO_OPEN, "w");
        fprintf(fp, "N, BLOCKSIZE, GRIDSIZE, MAX_DIGIT, GIPS, TIME_SEC\n");
    }
    fprintf(fp, "%f, %d, %d, %d, %f, %.5f\n", N, BLOCKSIZE, GRIDSIZE, MAX_DIGIT, GIPS / (time / 1000), time / 1000);
    fclose(fp);
}
/**
 * This functions test if the array is correctly sorted.
 * */
void TESTArray(int *array, int size) {
    for (int i = 1; i < size; i++)
        if (array[i - 1] > array[i]) {
            printf("\nERRORE NELL'ORDINAMENTO!\n");
            break;
        }
    printf("Ordinamento Corretto");
}
/**
 * This functions allocates all the resurces and launches all the kernel necessary
 * to sort the array
 * */
void radixSort(int *array, int size) {
    int significantDigit = 1;
    hipEvent_t start, stop;
    int threadCount;
    int blockCount;
    int pocket;
    int radix;
    int min, max;

    hipStream_t stream[MAXSM];

    for (int i = 0; i <= MAXSM; i++)
        hipStreamCreate(&stream[i]);
    threadCount = BLOCKSIZE;
    blockCount = GRIDSIZE;

    int max_digit_value;

    int *outputArray;
    int *inputArray;
    int *radixArray;
    int *bucketArray;
    int *indexArray;
    int *semiSortArray;
    int *blockBucketArray;
    int *g_maxdata;
    int *g_mindata;
    int *largestNum;
    int *smallestNum;

    int new_size_first = size / MAXSM;
    int my_size, offset = 0;
    int new_block_size;

    CUDA_CHECK(hipMalloc((void **)&inputArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&indexArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&g_maxdata, sizeof(int) * GRIDSIZE));
    CUDA_CHECK(hipMalloc((void **)&g_mindata, sizeof(int) * GRIDSIZE));

    CUDA_CHECK(hipMalloc((void **)&radixArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&outputArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&semiSortArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&bucketArray, sizeof(int) * RADIX));
    CUDA_CHECK(hipMalloc((void **)&blockBucketArray, sizeof(int) * RADIX * GRIDSIZE));

    hipMalloc((void **)&largestNum, sizeof(int));
    hipMalloc((void **)&smallestNum, sizeof(int));

    for (int j = 1; j <= MAXSM; j++) {
        hipMemcpyAsync(inputArray + new_size_first * (j - 1) + size % MAXSM, array + new_size_first * (j - 1) + size % MAXSM, new_size_first * sizeof(int), hipMemcpyHostToDevice, stream[j]);
    }

    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();
    // Create the channel between global and texture memory and bind them.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
    mycudaerror = hipBindTexture(0, texture_semiSortArray, semiSortArray, channelDesc);
    if (mycudaerror != hipSuccess) printf("can not bind semiSortArray to texture \n");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    reduceMaxMin<<<blockCount, threadCount>>>(inputArray, g_maxdata, g_mindata);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    reduceMaxMin_Service<<<1, BLOCKSIZE>>>(g_maxdata, g_mindata, largestNum, smallestNum);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }

    hipMemcpy(&max, largestNum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, smallestNum, sizeof(int), hipMemcpyDeviceToHost);

    int *bucket = (int *)malloc(RADIX * sizeof(int));
    int *CPUradixArray = (int *)malloc(size * sizeof(int));
    int *CPUindexArray = (int *)malloc(size * sizeof(int));

    // We add minimum number of the array with the maximum one in order to support also the sorting of the negative numbers
    max_digit_value = max - min;
    // We iterate on the number of digit contained in the max_digit_value
    while (max_digit_value / significantDigit > 0) {
        resetBucket<<<GRIDSIZE, RADIX>>>(blockBucketArray);
        resetBucket<<<1, RADIX>>>(bucketArray);
        for (int j = 1; j <= MAXSM; j++) {
            my_size = new_size_first;
            offset = new_size_first * (j - 1);

            new_block_size = (my_size - 1) / BLOCKSIZE + 1;

            histogramKernel<<<new_block_size, BLOCKSIZE, 0, stream[j]>>>(inputArray + offset, blockBucketArray + (j - 1) * new_block_size * RADIX, radixArray + offset, my_size, significantDigit, min);

            mycudaerror = hipGetLastError();
            if (mycudaerror != hipSuccess) {
                fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
                exit(1);
            }

            combineBucket<<<1, RADIX, 0, stream[j]>>>(blockBucketArray + (j - 1) * new_block_size * RADIX, bucketArray, new_block_size);

            mycudaerror = hipGetLastError();
            if (mycudaerror != hipSuccess) {
                fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
                exit(1);
            }
        }
        // This sections is dedicated to estabilishes the specific position at which collocate the number. (Replacing indexArrayKernel)
        hipMemcpy(CPUradixArray, radixArray, sizeof(int) * size, hipMemcpyDeviceToHost);
        hipMemcpy(bucket, bucketArray, sizeof(int) * RADIX, hipMemcpyDeviceToHost);
        for (int c = 0; c < size; c++) {
            radix = CPUradixArray[size - c - 1];
            pocket = --bucket[radix];
            CPUindexArray[size - c - 1] = pocket;
        }
        hipMemcpy(indexArray, CPUindexArray, sizeof(int) * size, hipMemcpyHostToDevice);

        hipDeviceSynchronize();

        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        for (int j = 1; j <= MAXSM; j++) {
            my_size = new_size_first;
            offset = new_size_first * (j - 1);

            new_block_size = (my_size - 1) / BLOCKSIZE + 1;

            semiSortKernel<<<new_block_size, BLOCKSIZE, 0, stream[j]>>>(inputArray + offset, semiSortArray, indexArray + offset, my_size, significantDigit);
            mycudaerror = hipGetLastError();
            if (mycudaerror != hipSuccess) {
                fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
                exit(1);
            }
        }
        hipDeviceSynchronize();
        for (int j = 1; j <= MAXSM; j++) {
            my_size = new_size_first;
            offset = new_size_first * (j - 1);

            new_block_size = (my_size - 1) / BLOCKSIZE + 1;
            copyKernel<<<new_block_size, BLOCKSIZE, 0, stream[j]>>>(inputArray + offset, offset, my_size);

            mycudaerror = hipGetLastError();
            if (mycudaerror != hipSuccess) {
                fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
                exit(1);
            }
        }

        significantDigit *= RADIX;
    }
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float transferTime;
    hipEventElapsedTime(&transferTime, start, stop);
    printf("CUDA Time = %.5f ms GIPS = %.5f MAX_DIGIT = %d BLOCKSIZE = %d dim=%d\n", transferTime, GIPS, MAX_DIGIT, BLOCKSIZE, size);
    make_csv(transferTime, size);
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(g_maxdata);
    hipFree(g_mindata);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);

    hipUnbindTexture(texture_semiSortArray);
}

int main() {
    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int *array = (int *)malloc(size * sizeof(int));
    int i;
    srand(time(NULL));

    for (i = 0; i < size; i++) {
        if (i % 2)
            array[i] = -(rand() % MAX_DIGIT);
        else
            array[i] = (rand() % MAX_DIGIT);
    }

    // printf("\nUnsorted List: ");
    // printArray(array, size);

    radixSort(array, size);

    if (TEST) {
        TESTArray(array, size);
    }
    // printf("\nSorted List:");
    // printArray(array, size);

    printf("\n");

    return 0;
}