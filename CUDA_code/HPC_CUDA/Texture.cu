#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define CUDA_CHECK(X)                                                     \
    {                                                                     \
        hipError_t _m_cudaStat = X;                                      \
        if (hipSuccess != _m_cudaStat) {                                 \
            fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
                    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#ifndef SIZE
#define SIZE 14155776
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 1024
#endif

#ifndef MAX_DIGIT
#define MAX_DIGIT 9999
#endif

#ifndef GIPS
#define GIPS 0
#endif

#ifndef TEST
#define TEST 0
#endif

#define GRIDSIZE ((SIZE - 1) / BLOCKSIZE + 1)
#define RADIX 10
#define FILE_TO_OPEN "Texture_measures.csv"

texture<int, 1> texture_radixArray;
/**
 * This device function is used to retrive values from the array
 * saved in the texture memory
 * */
__device__ int fetch_radixArrayElement(int value) {
    return tex1Dfetch(texture_radixArray, value);
}
/**
 *  This kernel will be launched on GRIDSIZE * BLOCKSIZE threads, in order to copy the values of the semiSortArray in inArray
 *
 * **/
__global__ void copyKernel(int *inArray, int *semiSortArray, int arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < arrayLength) {
        inArray[index] = semiSortArray[index];
    }
}
/**
 * This kernel will be launched on GRIDSIZE * BLOCKSIZE threads, so that each thread will calculate its local maximum and minimum value
 *
 * */
__global__ void reduceMaxMin(int *g_idata, int *g_maxdata, int *g_mindata) {
    __shared__ int smaxdata[(SIZE / GRIDSIZE)];
    __shared__ int smindata[(SIZE / GRIDSIZE)];
    int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    smaxdata[tid] = g_idata[i];
    smindata[tid] = g_idata[i];
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem

    if (tid == 0) {
        g_maxdata[blockIdx.x] = smaxdata[0];
        g_mindata[blockIdx.x] = smindata[0];
    }
}
/**
 * This kernel will be launched on 1 * BLOCKSIZE threads, so that a global
 * maximum and minimum value shared between all blocks will be calcualted
 * */
__global__ void reduceMaxMin_Service(int *g_maxdata, int *g_mindata, int *max, int *min) {
    __shared__ int smaxdata[(BLOCKSIZE)];
    __shared__ int smindata[(BLOCKSIZE)];
    int tid = threadIdx.x;
    smaxdata[tid] = g_maxdata[tid];
    smindata[tid] = g_mindata[tid];
    for (unsigned int s = 1; s < GRIDSIZE / BLOCKSIZE; s++) {
        int index = BLOCKSIZE * s + tid;
        if (smaxdata[tid] < g_maxdata[index])
            smaxdata[tid] = g_maxdata[index];
        if (smindata[tid] > g_mindata[index])
            smindata[tid] = g_mindata[index];
    }
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem
    if (tid == 0) {
        *max = smaxdata[0];
        *min = smindata[0];
    }
}
/**
 * This kernel will be launched on GRIDSIZE * BLOCKSIZE threads, so that each thread for
 * the specific significant digit of the assigned value will increase the frequencies of the digit
 * */
__global__ void histogramKernel(int *inArray, int *outArray, int *radixArray, int arrayLength, int significantDigit, int minElement) {
    __shared__ int inArrayShared[BLOCKSIZE];
    __shared__ int outArrayShared[RADIX];
    __shared__ int radixArrayShared[BLOCKSIZE];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread = threadIdx.x;
    int blockIndex = blockIdx.x * RADIX;

    int radix;
    int arrayElement;
    int i;

    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArrayShared[i] = 0;
        }
    }

    if (index < arrayLength) {
        inArrayShared[thread] = inArray[index];
    }

    __syncthreads();

    if (index < arrayLength) {
        arrayElement = inArrayShared[thread] - minElement;
        radix = ((arrayElement / significantDigit) % 10);
        radixArrayShared[thread] = radix;
        atomicAdd(&outArrayShared[radix], 1);
    }

    if (index < arrayLength) {
        radixArray[index] = radixArrayShared[thread];
    }
    __syncthreads();
    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[blockIndex + i] = outArrayShared[i];
        }
    }
}
/**
 * This kernel will be launched on 1 * RADIX threads, so that the array containing the frequencies
 * for each block is addictioned to that of the other blocks.
 * Then the value in each position of the resulting array is addictioned with the value in the previus one position.
 * */
__global__ void combineBucket(int *blockBucketArray, int *bucketArray) {
    __shared__ int bucketArrayShared[RADIX];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index] = 0;

    for (i = index; i < RADIX * GRIDSIZE; i = i + RADIX) {
        atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 1; i < RADIX; i++)
            bucketArrayShared[i] += bucketArrayShared[i - 1];
    }
    __syncthreads();
    bucketArray[index] = bucketArrayShared[index];
}
/**
 * This kernel will be launched on 1 * RADIX threads, so that each thread
 * takes care of one digit between 0-9 and estabilishes the specific position
 * at which collocate the number
 * */
__global__ void indexArrayKernel(int *bucketArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    int radix;
    int pocket;

    if (index < RADIX) {
        for (i = 0; i < arrayLength; i++) {
            radix = fetch_radixArrayElement(arrayLength - i - 1);
            if (radix == index) {
                pocket = --bucketArray[radix];
                indexArray[arrayLength - i - 1] = pocket;
            }
        }
    }
}
/**
 * This kernel will be launched on GRIDSIZE * BLOCKSIZE, so that a sorting
 * for the specific significantDigit between the numbers is made
 * */
__global__ void semiSortKernel(int *inArray, int *outArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;

    if (index < arrayLength) {
        arrayElement = inArray[index];
        arrayIndex = indexArray[index];
        outArray[arrayIndex] = arrayElement;
    }
}

/**
 * Print all the array
 * */
void printArray(int *array, int size) {
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}
/**
 * This functions makes the csv file with all the information needed
 * */
void make_csv(float time, float N) {
    FILE *fp;
    if (access(FILE_TO_OPEN, F_OK) == 0) {
        fp = fopen(FILE_TO_OPEN, "a");

    } else {
        fp = fopen(FILE_TO_OPEN, "w");
        fprintf(fp, "N, BLOCKSIZE, GRIDSIZE, MAX_DIGIT, GIPS, TIME_SEC\n");
    }
    fprintf(fp, "%f, %d, %d, %d, %f, %.5f\n", N, BLOCKSIZE, GRIDSIZE, MAX_DIGIT, GIPS / (time / 1000), time / 1000);
    fclose(fp);
}

/**
 * This functions test if the array is correctly sorted.
 * */
void TESTArray(int *array, int size) {
    for (int i = 1; i < size; i++)
        if (array[i - 1] > array[i]) {
            printf("\nERRORE NELL'ORDINAMENTO!\n");
            break;
        }
    printf("Ordinamento Corretto");
}
/**
 * This functions allocates all the resurces and launches all the kernel necessary
 * to sort the array
 * */
void radixSort(int *array, int size) {
    int significantDigit = 1;
    hipEvent_t start, stop;
    int threadCount;
    int blockCount;

    int min, max;

    threadCount = BLOCKSIZE;
    blockCount = GRIDSIZE;

    int *outputArray;
    int *inputArray;
    int *radixArray;
    int *bucketArray;
    int *indexArray;
    int *semiSortArray;
    int *blockBucketArray;
    int *g_maxdata;
    int *g_mindata;
    int *largestNum;
    int *smallestNum;
    CUDA_CHECK(hipMalloc((void **)&inputArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&indexArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&g_maxdata, sizeof(int) * GRIDSIZE));
    CUDA_CHECK(hipMalloc((void **)&g_mindata, sizeof(int) * GRIDSIZE));

    CUDA_CHECK(hipMalloc((void **)&radixArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&outputArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&semiSortArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&bucketArray, sizeof(int) * RADIX));
    CUDA_CHECK(hipMalloc((void **)&blockBucketArray, sizeof(int) * RADIX * GRIDSIZE));

    hipMemcpy(inputArray, array, sizeof(int) * size, hipMemcpyHostToDevice);

    // Create the channel between global and texture memory and bind them.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
    hipError_t errt = hipBindTexture(0, texture_radixArray, radixArray, channelDesc);
    if (errt != hipSuccess) printf("can not bind radixArray to texture \n");

    int max_digit_value;
    hipMalloc((void **)&largestNum, sizeof(int));
    hipMalloc((void **)&smallestNum, sizeof(int));

    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    reduceMaxMin<<<blockCount, threadCount>>>(inputArray, g_maxdata, g_mindata);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    reduceMaxMin_Service<<<1, BLOCKSIZE>>>(g_maxdata, g_mindata, largestNum, smallestNum);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }

    hipMemcpy(&max, largestNum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, smallestNum, sizeof(int), hipMemcpyDeviceToHost);

    // We add minimum number of the array with the maximum one in order to support also the sorting of the negative numbers
    max_digit_value = max - min;
    // We iterate on the number of digit contained in the max_digit_value
    while (max_digit_value / significantDigit > 0) {
        int bucket[RADIX] = {0};
        hipMemcpy(bucketArray, bucket, sizeof(int) * RADIX, hipMemcpyHostToDevice);

        histogramKernel<<<blockCount, threadCount>>>(inputArray, blockBucketArray, radixArray, size, significantDigit, min);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        combineBucket<<<1, RADIX>>>(blockBucketArray, bucketArray);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        indexArrayKernel<<<1, RADIX>>>(bucketArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        semiSortKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        copyKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, size);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        significantDigit *= RADIX;
    }
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float transferTime;
    hipEventElapsedTime(&transferTime, start, stop);
    printf("CUDA Time = %.5f ms GIPS = %.5f MAX_DIGIT = %d BLOCKSIZE = %d dim=%d\n", transferTime, GIPS, MAX_DIGIT, BLOCKSIZE, size);
    make_csv(transferTime, size);
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(g_maxdata);
    hipFree(g_mindata);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);

    hipUnbindTexture(texture_radixArray);
}

int main() {
    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int *array = (int *)malloc(size * sizeof(int));
    int i;
    srand(time(NULL));

    for (i = 0; i < size; i++) {
        if (i % 2)
            array[i] = -(rand() % MAX_DIGIT);
        else
            array[i] = (rand() % MAX_DIGIT);
    }

    // printf("\nUnsorted List: ");
    // printArray(array, size);

    radixSort(array, size);

    if (TEST) {
        TESTArray(array, size);
    }

    // printf("\nSorted List:");
    // printArray(array, size);

    printf("\n");

    return 0;
}